#include "hip/hip_runtime.h"
#include <wb.h>

//@@ Complete this function
__global__ void d_axpy(const float * x, float * y, float a, int len)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if(i<len)
     y[i] = a* x[i] + y[i];
}

//Sequential Implementation
void h_axpy(const float * x, float * y, float a, int len) 
{
    for (int i = 0; i < len; i++) 
    {
        y[i] += a * x[i];
    }
}

int main(int argc, char **argv) {
  wbArg_t args;
  int inputLength;
  float *h_x;
  float *h_y;
  float a;
  float *d_x;
  float *d_y;

  args = wbArg_read(argc, argv);

  wbTime_start(Generic, "Importing data and creating memory on host");
  h_x =
      (float *)wbImport(wbArg_getInputFile(args, 0), &inputLength);
  h_y =
      (float *)wbImport(wbArg_getInputFile(args, 1), &inputLength);
  {
      int aLength;
      float * pA = (float *)wbImport(wbArg_getInputFile(args, 2), &aLength);
      a = *pA;

      free(pA);
  }

  wbTime_stop(Generic, "Importing data and creating memory on host");
  
  wbLog(TRACE, "The input length is ", inputLength);
  
  //@@ Allocate memory
  hipMalloc((void **)&d_x,inputLength*sizeof(float));
  hipMalloc((void **)&d_y,inputLength*sizeof(float));
  
  // Copy memory to the GPU
  hipMemcpy(d_x,h_x,inputLength*sizeof(float),hipMemcpyHostToDevice);
  hipMemcpy(d_y,h_y,inputLength*sizeof(float),hipMemcpyHostToDevice);
  
  //@@ Initialize the grid and block dimensions here  
   dim3 blockDim(32);
   dim3 gridDim(ceil(((float)inputLength)/((float)blockDim.x)));
  
  //@@ Performing CUDA compute
  wbTime_start(GPU, "Doing GPU Computation (memory + compute)");
  d_axpy<<<gridDim,blockDim>>>(d_x,d_y,a,inputLength);
  hipDeviceSynchronize();
  wbTime_stop(GPU, "Doing GPU Computation (memory + compute)");
  
  //@@ Copy the GPU memory back to the CPU here  
  hipMemcpy(h_y,d_y,inputLength*sizeof(float),hipMemcpyDeviceToHost);
  
  //@@ Free the GPU memory here 
   hipFree(d_x);
   hipFree(d_y);

  // Verify correctness of the results
  wbLog(TRACE, "The result is ", h_y[0]);
  wbLog(TRACE, "The result is ", h_y[1]);
  wbLog(TRACE, "The result is ", h_y[2]);
  wbLog(TRACE, "The result is ", h_y[3]);
  wbLog(TRACE, "The result is ", h_y[4]);
  wbLog(TRACE, "The result is ", h_y[5]);
  wbLog(TRACE, "The result is ", h_y[6]);
  wbLog(TRACE, "The result is ", h_y[7]);
  wbLog(TRACE, "The result is ", h_y[8]);
  wbLog(TRACE, "The result is ", h_y[9]);
  
   wbSolution(args, h_y, inputLength);


  //Free the host memory
  free(h_x);
  free(h_y);

  return 0;
}
