#include "hip/hip_runtime.h"
#include <wb.h>

// Error check
#define cudaCheckError() {                                                                       \
        hipError_t e=hipGetLastError();                                                        \
        if(e!=hipSuccess) {                                                                     \
            printf("Cuda failure %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(e));        \
            exit(EXIT_FAILURE);                                                                  \
        }                                                                                        \
    }
    
    

__global__ void vecAdd(float *in1, float *in2, float *out, int len) 
{
  //@@ Insert code to implement vector addition here
    
  int index = blockIdx.x*blockDim.x + threadIdx.x;
  
  if(index<len)
     out[index] = in1[index] + in2[index];
     
}


int main(int argc, char **argv) 
{

  wbArg_t args;    // used by library, to pass input variables(in1, in2, out) to the executable
  int inputLength;
  float *hostInput1;
  float *hostInput2;
  float *hostOutput;
  float *deviceInput1;
  float *deviceInput2;
  float *deviceOutput;

  args = wbArg_read(argc, argv);  // read data from the files (input1, input2, output)
  
  

  wbTime_start(Generic, "Importing data and creating memory on host");//-----------------------------------------// 
  //@@ Extract data and store in host variables (allocate and populate memory with values)
  // pass by reference for 'inputLength' so variable can be modified inside the function based on the Dataset used
   hostInput1 =
      (float *)wbImport(wbArg_getInputFile(args, 0), &inputLength);// -i (path0 on terminal)
   hostInput2 =
      (float *)wbImport(wbArg_getInputFile(args, 1), &inputLength);// -i (path1 on terminal)
   hostOutput = (float *)malloc(inputLength * sizeof(float));
  wbTime_stop(Generic, "Importing data and creating memory on host");//-----------------------------------------//
  
  

  wbLog(TRACE, "The input length is ", inputLength);
  
  

  wbTime_start(GPU, "Allocating GPU memory.");//--------------------------------------------------// 
  //@@ Allocate GPU memory here
   hipMalloc((void **)&deviceInput1,inputLength*sizeof(float)); // predefined vars, so use pass by reference
   hipMalloc((void **)&deviceInput2,inputLength*sizeof(float));
   hipMalloc((void **)&deviceOutput,inputLength*sizeof(float));  
  wbTime_stop(GPU, "Allocating GPU memory.");//----------------------------------------------------// 



  wbTime_start(GPU, "Copying input memory to the GPU.");//------------------------------------------// 
  //@@ Copy memory to the GPU here
   hipMemcpy(deviceInput1,hostInput1,inputLength*sizeof(float),hipMemcpyHostToDevice);
   hipMemcpy(deviceInput2,hostInput2,inputLength*sizeof(float),hipMemcpyHostToDevice);
  wbTime_stop(GPU, "Copying input memory to the GPU.");//------------------------------------------// 



  //@@ Initialize the grid and block dimensions here
   dim3 blockDim(32);
   dim3 gridDim(ceil(((float)inputLength)/((float)blockDim.x)));
  


  wbTime_start(Compute, "Performing CUDA computation");//--------------------------------------------// 
  //@@ Launch the GPU Kernel here
   vecAdd<<<gridDim,blockDim>>>(deviceInput1,deviceInput2,deviceOutput,inputLength);
   hipDeviceSynchronize(); 
  wbTime_stop(Compute, "Performing CUDA computation");//---------------------------------------------// 
  


  wbTime_start(Copy, "Copying output memory to the CPU");//------------------------------------------// 
  //@@ Copy the GPU memory back to the CPU here 
   hipMemcpy(hostOutput,deviceOutput,inputLength*sizeof(float),hipMemcpyDeviceToHost);
  wbTime_stop(Copy, "Copying output memory to the CPU");//-------------------------------------------// 



  wbTime_start(GPU, "Freeing GPU Memory");//--------------------------------------------------------// 
   //@@ Free the GPU memory here
   hipFree(deviceInput1);
   hipFree(deviceInput2);
   hipFree(deviceOutput);
  wbTime_stop(GPU, "Freeing GPU Memory");//---------------------------------------------------------// 


  wbSolution(args, hostOutput, inputLength);//Compares with output in dataset
  

  free(hostInput1);
  free(hostInput2);
  free(hostOutput);

  cudaCheckError();
  return 0; 
  
}
